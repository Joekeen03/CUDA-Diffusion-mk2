
#include <hip/hip_runtime.h>
#include <iostream>

namespace DiffusionSimulation {
    /**
     * TODO:
     *  -Check cudaMallocPitch vs. cudaMallocManaged (for speed)
     *  -Check breaking if-else possibilities into separate kernels
     *  -See if I can split the threads that handle the boundaries into their own warps.
     *  -Error checking - validate that the diffusion kernel runs correctly, with initially uniform
     *      concentration, and a "drop" in the middle
     *  -Get diffusion simulation processing more than one time step, by swapping the arrays each time step
     *      *Simple - synch with device, perform swap on CPU
     *      *Maybe harder - make a kernel that performs the swap, so it all happens on the GPU.
     *  -Set up diffusion simulation to output to screen
     *  -Something else?
    */
    __global__
    void Diffusion(float** current, float** next, float xN, float yN, float dx, float dy, float dt, float D) {
        const float xFactor = D*dt*dy/dx;
        const float yFactor = D*dt*dx/dy;
        const int xIndex = blockIdx.x*blockDim.x + threadIdx.x;
        const int yIndex = blockIdx.y*blockDim.y + threadIdx.y;
        // Assuming row-major order, 0,0 is at the bottom-left corner
        const float oldVal = current[yIndex][xIndex];
        float yDiff = 0.0f;
        if (yIndex < (yN-1) && yIndex > 0) {
            yDiff = (current[yIndex+1][xIndex]-2*current[yIndex][xIndex]+current[yIndex-1][xIndex])/(dy*dy);
        } else {
            if (yIndex == 0) {
                yDiff = (-current[yIndex+2][xIndex]+8*current[yIndex+2][xIndex]-7*current[yIndex][xIndex])/(2*dy*dy);
            } else {
                yDiff = (-current[yIndex-2][xIndex]+8*current[yIndex-1][xIndex]-7*current[yIndex][xIndex])/(2*dy*dy);
            }
        }
        float xDiff = 0.0f;
        if (xIndex < (xN-1) && xIndex > 0) {
            // (d^2/dx^2)(f(x, y, t)) = (-f(x+2dx)+8*f(x-dx)-7*f(x))/(2dx^2) + O(h^2)
            xDiff = (current[yIndex][xIndex+1]-2*current[yIndex][xIndex]+current[yIndex][xIndex-1])/(dx*dx);
        } else {
            if (xIndex == 0) {
                xDiff = (-current[yIndex][xIndex+2]+8*current[yIndex][xIndex+1]-7*current[yIndex][xIndex])/(2*dx*dx);
            } else {
                xDiff = (-current[yIndex][xIndex-2]+8*current[yIndex][xIndex-1]-7*current[yIndex][xIndex])/(2*dx*dx);
            }
        }
        float phiDiff = -D*(xDiff+yDiff)*dt;
        next[yIndex][xIndex] = phiDiff;
    }

    void RunDiffusion() {
        float xLength = 1000;
        float yLength = 1000;
        
        float D = 1.0f;

        float dx = 0.1f;
        float dy = 0.1;
        float dt = 0.1f;

        int xSize = (int) ceil(xLength/dx);
        int ySize = (int) ceil(yLength/dy);

        float **current;
        float **next;

        hipMallocManaged(&current, xLength*yLength*sizeof(float));
        hipMallocManaged(&next, xLength*yLength*sizeof(float));
        for (size_t y = 0; y < ySize; y++) {
            for (size_t x = 0; x < xSize; x++) {
                current[y][x] = 0.0f;
                next[y][x] = 0.0f;
            }
        }
        
        float initialTotal = 0.0f;
        for (size_t y = 0; y < ySize; y++) {
            for (size_t x = 0; x < xSize; x++) {
                initialTotal += current[y][x];
            }
        }

        Diffusion<<<1, 1>>>(current, next, xSize, ySize, dx, dy, dt, D);
        hipDeviceSynchronize();
        
        float finalTotal = 0.0f;
        for (size_t y = 0; y < ySize; y++) {
            for (size_t x = 0; x < xSize; x++) {
                finalTotal += current[y][x];
            }
        }

        printf("Difference: %f", finalTotal-initialTotal);

        hipFree(current);
        hipFree(next);
    }
}